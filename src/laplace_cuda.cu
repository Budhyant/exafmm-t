#include "hip/hip_runtime.h"
#include "exafmm_t.h"
#include "profile.h"
#include "laplace_cuda.h"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace exafmm_t {
  __global__
  void P2M_potential_p2p_kernel(int *leafs_idx, int *d_nodes_pt_src_idx, real_t *d_nodes_coord, real_t *d_nodes_pt_src, real_t *d_checkCoord, real_t *d_upward_equiv, real_t *d_r, real_t *d_leaf_xyz) {
    int i = blockIdx.x;
    int t = threadIdx.x;
    const real_t COEF = 1.0/(2*4*M_PI);
    real_t tx = d_checkCoord[3*t+0]*d_r[i]+d_leaf_xyz[3*i+0];
    real_t ty = d_checkCoord[3*t+1]*d_r[i]+d_leaf_xyz[3*i+1];
    real_t tz = d_checkCoord[3*t+2]*d_r[i]+d_leaf_xyz[3*i+2];
    real_t tv = 0;
    int src_cnt = (d_nodes_pt_src_idx[leafs_idx[i]+1]-d_nodes_pt_src_idx[leafs_idx[i]]);
    int leaf_pt_src_idx = d_nodes_pt_src_idx[leafs_idx[i]];
    int leaf_coord_idx = 3*leaf_pt_src_idx;
    for(int s=0; s<src_cnt; s++) {
      real_t sx = d_nodes_coord[leaf_coord_idx+3*s+0]-tx;
      real_t sy = d_nodes_coord[leaf_coord_idx+3*s+1]-ty;
      real_t sz = d_nodes_coord[leaf_coord_idx+3*s+2]-tz;
      real_t sv = d_nodes_pt_src[leaf_pt_src_idx+s];
      real_t r2 = sx*sx + sy*sy + sz*sz;;
      if (r2 != 0) {
        real_t invR = rsqrtf(r2);
        tv += invR * sv;
      }
    }
    tv *= COEF;
    d_upward_equiv[leafs_idx[i]*blockDim.x+t] += tv;
  }

  __global__
  void M2MKernel(real_t *d_upward_equiv, real_t *d_buffer, int *d_parent_by_level_idx) {
    atomicAdd(&d_upward_equiv[d_parent_by_level_idx[blockIdx.x]*blockDim.x+threadIdx.x], d_buffer[blockIdx.x*blockDim.x+threadIdx.x]);
  }

 __global__
  void gradientP2PKernel(int *d_leafs_idx, int *d_nodes_pt_src_idx, int *d_P2Plists, int *d_P2Plists_idx, real_t *d_nodes_coord, real_t *d_nodes_pt_src, real_t *d_trg_val) {
    const real_t COEFP = 1.0/(2*4*M_PI);
    const real_t COEFG = -1.0/(4*2*2*6*M_PI);

    int first_trg_coord_idx = 3*d_nodes_pt_src_idx[d_leafs_idx[blockIdx.x]];
    int trg_coord_size = 3*(d_nodes_pt_src_idx[d_leafs_idx[blockIdx.x]+1] - d_nodes_pt_src_idx[d_leafs_idx[blockIdx.x]]);
    int first_trg_val_idx = 4*first_trg_coord_idx/3;
    if (threadIdx.x < trg_coord_size/3) {
      real_t tx = d_nodes_coord[first_trg_coord_idx+3*threadIdx.x+0];
      real_t ty = d_nodes_coord[first_trg_coord_idx+3*threadIdx.x+1];
      real_t tz = d_nodes_coord[first_trg_coord_idx+3*threadIdx.x+2];
      real_t tv0=0;
      real_t tv1=0;
      real_t tv2=0;
      real_t tv3=0;

      int first_p2plist_idx = d_P2Plists_idx[blockIdx.x];
      int P2Plist_size = d_P2Plists_idx[blockIdx.x+1] - d_P2Plists_idx[blockIdx.x];
      for(int j=0; j<P2Plist_size; j++) {
        int src_idx = d_P2Plists[first_p2plist_idx+j];
        int first_src_coord_idx = 3*d_nodes_pt_src_idx[src_idx];
        int src_coord_size = 3*(d_nodes_pt_src_idx[src_idx+1] - d_nodes_pt_src_idx[src_idx]);
        int first_src_val_idx = d_nodes_pt_src_idx[src_idx];
        for(int k=0; k<src_coord_size/3; k ++) {
          real_t sx = d_nodes_coord[first_src_coord_idx + 3*k + 0] - tx;
          real_t sy = d_nodes_coord[first_src_coord_idx + 3*k + 1] - ty;
          real_t sz = d_nodes_coord[first_src_coord_idx + 3*k + 2] - tz;
          real_t r2 = sx*sx + sy*sy + sz*sz;
          real_t sv = d_nodes_pt_src[first_src_val_idx+k];
          if (r2 != 0) {
            real_t invR = rsqrtf(r2);
            real_t invR3 = invR*invR*invR;
            tv0 += invR*sv;
            sv *= invR3;
            tv1 += sv*sx;
            tv2 += sv*sy;
            tv3 += sv*sz;
          }
        }
      }
      tv0 *= COEFP;
      tv1 *= COEFG;
      tv2 *= COEFG;
      tv3 *= COEFG;
      d_trg_val[first_trg_val_idx+4*threadIdx.x+0] += tv0;
      d_trg_val[first_trg_val_idx+4*threadIdx.x+1] += tv1;
      d_trg_val[first_trg_val_idx+4*threadIdx.x+2] += tv2;
      d_trg_val[first_trg_val_idx+4*threadIdx.x+3] += tv3;
    }
  }
   
  __global__
  void hadmard_kernel(int *d_M2Ltargets_idx, hipfftComplex *d_up_equiv_fft, hipfftComplex *d_dw_equiv_fft, int *d_M2LRelPos_start_idx, int *d_index_in_up_equiv_fft, int *d_M2LRelPoss, real_t *d_mat_M2L_Helper, int n3_, int BLOCKS) {
    int i = blockIdx.x;
    int k = threadIdx.x;
    int M2LRelPos_size = d_M2LRelPos_start_idx[i+1]-d_M2LRelPos_start_idx[i];
    for(int j=0; j <M2LRelPos_size; j++) {
      int relPosidx = d_M2LRelPoss[d_M2LRelPos_start_idx[i]+j];
      real_t *kernel = &d_mat_M2L_Helper[relPosidx*2*n3_];
      hipfftComplex *equiv = &d_up_equiv_fft[d_index_in_up_equiv_fft[d_M2LRelPos_start_idx[i]+j]*n3_];
      hipfftComplex *check = &d_dw_equiv_fft[i*n3_];
      int real = 2*k+0;
      int imag = 2*k+1;
      check[k].x += kernel[real]*equiv[k].x - kernel[imag]*equiv[k].y;
      check[k].y += kernel[real]*equiv[k].y + kernel[imag]*equiv[k].x;
    }
  }

  void cuda_init_drivers() {
    hipFree(0);
}
  
  void P2MGPU(std::vector<int> &leafs_idx, std::vector<real_t> &nodes_coord, std::vector<real_t> &nodes_pt_src, std::vector<int> &nodes_pt_src_idx, std::vector<real_t> &checkCoord, int trg_cnt, RealVec &upward_equiv, std::vector<real_t> &r, std::vector<real_t> &leaf_xyz, int leafs_size, int ncrit) {
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);

    int BLOCKS = leafs_size;
    int THREADS = trg_cnt/3; 
    int *d_nodes_pt_src_idx, *d_leafs_idx;
    real_t *d_nodes_coord, *d_nodes_pt_src, *d_checkCoord, *d_upward_equiv, *d_r, *d_leaf_xyz, *d_M2M_V, *d_buffer, *d_M2M_U;

    hipMalloc(&d_nodes_pt_src_idx, sizeof(int)*nodes_pt_src_idx.size());
    hipMalloc(&d_nodes_coord, sizeof(real_t)*nodes_coord.size());
    hipMalloc(&d_nodes_pt_src, sizeof(real_t)*nodes_pt_src.size());
    hipMalloc(&d_checkCoord, sizeof(real_t)*checkCoord.size());
    hipMalloc(&d_upward_equiv, sizeof(real_t)*upward_equiv.size());
    hipMalloc(&d_r, sizeof(real_t)*r.size());
    hipMalloc(&d_leaf_xyz, sizeof(real_t)*leaf_xyz.size());
    hipMalloc(&d_M2M_V, sizeof(real_t)*M2M_V.size());
    hipMalloc(&d_buffer, sizeof(real_t)*leafs_idx.size()*NSURF);
    hipMalloc(&d_M2M_U, sizeof(real_t)*M2M_U.size());
    hipMalloc(&d_leafs_idx, sizeof(int)*leafs_idx.size());

    hipMemcpy(d_leafs_idx, &leafs_idx[0], sizeof(int)*leafs_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_M2M_U, &M2M_U[0], sizeof(real_t)*M2M_U.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_M2M_V, &M2M_V[0], sizeof(real_t)*M2M_V.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_pt_src_idx, &nodes_pt_src_idx[0], sizeof(int)*nodes_pt_src_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_coord, &nodes_coord[0], sizeof(real_t)*nodes_coord.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_pt_src, &nodes_pt_src[0], sizeof(real_t)*nodes_pt_src.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_checkCoord, &checkCoord[0], sizeof(real_t)*checkCoord.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_upward_equiv, &upward_equiv[0], sizeof(real_t)*upward_equiv.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_r, &r[0], sizeof(real_t)*r.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_leaf_xyz, &leaf_xyz[0], sizeof(real_t)*leaf_xyz.size(), hipMemcpyHostToDevice);
    Profile::Tic("general",true);
    P2M_potential_p2p_kernel<<<BLOCKS, THREADS>>>(d_leafs_idx, d_nodes_pt_src_idx, d_nodes_coord, d_nodes_pt_src, d_checkCoord, d_upward_equiv, d_r, d_leaf_xyz);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    real_t alpha=1.0, beta=0.0;
    real_t **M2M_V_p = 0, **upward_equiv_p = 0, **buffer_p = 0, **M2M_U_p;
    M2M_V_p = (real_t**)malloc(leafs_size * sizeof(real_t*));
    upward_equiv_p = (real_t**)malloc(leafs_size * sizeof(real_t*));
    buffer_p = (real_t**)malloc(leafs_size * sizeof(real_t*));
    M2M_U_p = (real_t**)malloc(leafs_size * sizeof(real_t*));
    for(int i = 0; i < leafs_size; i++){
      M2M_V_p[i] = d_M2M_V;
      upward_equiv_p[i] = d_upward_equiv + leafs_idx[i]*NSURF;
      buffer_p[i] = d_buffer + i*NSURF;
      M2M_U_p[i] = d_M2M_U;
    }
    real_t **d_M2M_V_p = 0, **d_upward_equiv_p = 0, **d_buffer_p = 0, **d_M2M_U_p=0;
    hipMalloc(&d_M2M_V_p, leafs_size*sizeof(real_t*));
    hipMalloc(&d_upward_equiv_p, leafs_size*sizeof(real_t*));
    hipMalloc(&d_buffer_p, leafs_size*sizeof(real_t*));
    hipMalloc(&d_M2M_U_p, leafs_size*sizeof(real_t*));
    hipMemcpy(d_M2M_V_p, M2M_V_p, sizeof(real_t*)*leafs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_upward_equiv_p, upward_equiv_p, sizeof(real_t*)*leafs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_buffer_p, buffer_p, sizeof(real_t*)*leafs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_M2M_U_p, M2M_U_p, sizeof(real_t*)*leafs_size, hipMemcpyHostToDevice);

    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NSURF, 1, NSURF, &alpha, (const float**)d_M2M_V_p, NSURF, (const float**)d_upward_equiv_p, NSURF, &beta, d_buffer_p, NSURF, leafs_size);
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NSURF, 1, NSURF, &alpha, (const float**)d_M2M_U_p, NSURF, (const float**)d_buffer_p, NSURF, &beta, d_upward_equiv_p, NSURF, leafs_size);
    Profile::Toc();
    hipMemcpy(&upward_equiv[0], d_upward_equiv, sizeof(real_t)*upward_equiv.size(), hipMemcpyDeviceToHost);
    hipFree(d_nodes_pt_src_idx);
    hipFree(d_leafs_idx);
    hipFree(d_nodes_coord);
    hipFree(d_nodes_pt_src);
    hipFree(d_checkCoord);
    hipFree(d_upward_equiv);
    hipFree(d_r);
    hipFree(d_leaf_xyz);
    hipFree(d_M2M_U);
    hipFree(d_M2M_V);
    hipFree(d_buffer);
    hipblasDestroy(handle);
  }

void M2MGPU(RealVec &upward_equiv, std::vector<std::vector<int>> &nodes_by_level_idx, std::vector<std::vector<int>> &parent_by_level_idx, std::vector<std::vector<int>> &octant_by_level_idx) {
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    real_t *d_mat_M2M, *d_upward_equiv;
    hipMalloc(&d_upward_equiv, sizeof(real_t)*upward_equiv.size());
    hipMalloc(&d_mat_M2M, sizeof(real_t)*mat_M2M.size());    
    
    hipMemcpy(d_mat_M2M, &mat_M2M[0], sizeof(real_t)*mat_M2M.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_upward_equiv, &upward_equiv[0], sizeof(real_t)*upward_equiv.size(), hipMemcpyHostToDevice);
    for(int i=nodes_by_level_idx.size()-1;i>=0;i--) {
      real_t *d_buffer;
      
      float **AList = 0, **BList = 0, **CList = 0;
      AList = (real_t**)malloc(nodes_by_level_idx[i].size() * sizeof(real_t*));
      BList = (real_t**)malloc(nodes_by_level_idx[i].size() * sizeof(real_t*));
      CList = (real_t**)malloc(nodes_by_level_idx[i].size() * sizeof(real_t*));
      hipMalloc(&d_buffer, sizeof(real_t)*NSURF*nodes_by_level_idx[i].size());
      for(int j = 0; j < nodes_by_level_idx[i].size(); j++){
          AList[j] = d_upward_equiv + nodes_by_level_idx[i][j]*NSURF;
          BList[j] = d_mat_M2M + octant_by_level_idx[i][j]*NSURF*NSURF;
          CList[j] = d_buffer + j*NSURF;
      }
      real_t **d_AList, **d_BList, **d_CList;
      hipMalloc(&d_AList, nodes_by_level_idx[i].size() * sizeof(real_t*));
      hipMalloc(&d_BList, nodes_by_level_idx[i].size() * sizeof(real_t*));
      hipMalloc(&d_CList, nodes_by_level_idx[i].size() * sizeof(real_t*));

      hipMemcpy(d_CList, CList, sizeof(real_t*)*nodes_by_level_idx[i].size(), hipMemcpyHostToDevice);
      hipMemcpy(d_BList, BList, sizeof(real_t*)*nodes_by_level_idx[i].size(), hipMemcpyHostToDevice);
      hipMemcpy(d_AList, AList, sizeof(real_t*)*nodes_by_level_idx[i].size(), hipMemcpyHostToDevice);
      real_t alpha=1.0, beta=0.0;
      hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NSURF, 1, NSURF, &alpha, (const float**)d_BList, NSURF, (const float**)d_AList, NSURF, &beta, d_CList, NSURF, nodes_by_level_idx[i].size());
      int *d_parent_by_level_idx;
      hipMalloc(&d_parent_by_level_idx, parent_by_level_idx[i].size() * sizeof(int));
      hipMemcpy(d_parent_by_level_idx, &parent_by_level_idx[i][0], sizeof(int)*parent_by_level_idx[i].size(), hipMemcpyHostToDevice);
      M2MKernel<<<parent_by_level_idx[i].size(), NSURF>>>(d_upward_equiv, d_buffer, d_parent_by_level_idx);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      hipFree(d_buffer);
      hipFree(d_parent_by_level_idx);
    }
    hipMemcpy(&upward_equiv[0], d_upward_equiv, sizeof(real_t)*upward_equiv.size(), hipMemcpyDeviceToHost);
    hipFree(d_upward_equiv);
    hipFree(d_mat_M2M);
  }

  void P2PGPU(std::vector<int> &leafs_idx, std::vector<real_t> nodes_coord, std::vector<real_t> nodes_pt_src, std::vector<int> nodes_pt_src_idx, std::vector<int> P2Plists, std::vector<int> P2Plists_idx, std::vector<real_t> &trg_val, int leafs_size, int ncrit) {
    int BLOCKS = leafs_size;
    int THREADS = ncrit;

    int *d_nodes_pt_src_idx, *d_P2Plists, *d_P2Plists_idx, *d_leafs_idx;
    real_t *d_nodes_coord, *d_nodes_pt_src, *d_trg_val;
    
    hipMalloc(&d_leafs_idx, sizeof(int)*leafs_idx.size());
    hipMalloc(&d_nodes_pt_src_idx, sizeof(int)*nodes_pt_src_idx.size());
    hipMalloc(&d_P2Plists, sizeof(int)*P2Plists.size());
    hipMalloc(&d_P2Plists_idx, sizeof(int)*P2Plists_idx.size());
    hipMalloc(&d_nodes_coord, sizeof(real_t)*nodes_coord.size());
    hipMalloc(&d_nodes_pt_src, sizeof(real_t)*nodes_pt_src.size());
    hipMalloc(&d_trg_val, sizeof(real_t)*trg_val.size());
    
    Profile::Tic("host to device", true);
    hipMemcpy(d_leafs_idx, &leafs_idx[0], sizeof(int)*leafs_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_pt_src_idx, &nodes_pt_src_idx[0], sizeof(int)*nodes_pt_src_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_P2Plists, &P2Plists[0], sizeof(int)*P2Plists.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_P2Plists_idx, &P2Plists_idx[0], sizeof(int)*P2Plists_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_coord, &nodes_coord[0], sizeof(real_t)*nodes_coord.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_pt_src, &nodes_pt_src[0], sizeof(real_t)*nodes_pt_src.size(), hipMemcpyHostToDevice);
    Profile::Toc();
    Profile::Tic("gpu kernel", true);
    gradientP2PKernel<<<BLOCKS, THREADS>>>(d_leafs_idx, d_nodes_pt_src_idx, d_P2Plists, d_P2Plists_idx, d_nodes_coord, d_nodes_pt_src, d_trg_val);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    Profile::Toc();
    Profile::Tic("device to host", true);
    hipMemcpy(&trg_val[0], d_trg_val, sizeof(real_t)*trg_val.size(), hipMemcpyDeviceToHost);
    Profile::Toc();
    hipFree(d_leafs_idx);
    hipFree(d_nodes_coord);
    hipFree(d_nodes_pt_src);
    hipFree(d_P2Plists_idx);
    hipFree(d_P2Plists);
    hipFree(d_nodes_pt_src_idx);
    hipFree(d_trg_val);
  }

  hipfftComplex *FFT_UpEquiv_GPU(int M2Lsources_idx_size, AlignedVec &up_equiv) {
    int n1 = MULTIPOLE_ORDER * 2;
    int n3_ = n1 * n1 * (n1 / 2 + 1);
    int dims[] = {n1,n1,n1};
    hipfftHandle plan_up_equiv;
    hipfftPlanMany(&plan_up_equiv, 3, dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C, M2Lsources_idx_size); // first call to cufft will always take time so this is why ill leave it out from the timing
    hipfftComplex *d_up_equiv_fft;
    real_t *d_up_equiv;
    hipMalloc(&d_up_equiv, sizeof(real_t)*up_equiv.size());
    hipMalloc(&d_up_equiv_fft, sizeof(hipfftComplex)*M2Lsources_idx_size*n3_);
    hipMemcpy(d_up_equiv, &up_equiv[0], sizeof(real_t)*up_equiv.size(), hipMemcpyHostToDevice);
    hipfftExecR2C(plan_up_equiv, &d_up_equiv[0], &d_up_equiv_fft[0]);
    hipfftDestroy(plan_up_equiv);
    hipFree(d_up_equiv);
    return &d_up_equiv_fft[0];
  }

  std::vector<real_t> FFT_Check2Equiv_GPU(hipfftComplex *d_dw_equiv_fft, int M2Ltargets_idx_size) {
    int n1 = MULTIPOLE_ORDER * 2;
    int n3 = n1 * n1 * n1;
    int dims[] = {n1,n1,n1};
    
    real_t *d_dnCheck;
    hipMalloc(&d_dnCheck, sizeof(real_t)*M2Ltargets_idx_size*n3);
    hipfftHandle plan_check_equiv;
    hipfftPlanMany(&plan_check_equiv, 3, dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R, M2Ltargets_idx_size);
    hipfftExecC2R(plan_check_equiv, &d_dw_equiv_fft[0], &d_dnCheck[0]);
    hipfftDestroy(plan_check_equiv);
    std::vector<real_t> dnCheck(M2Ltargets_idx_size*n3);
    hipMemcpy(&dnCheck[0], d_dnCheck, sizeof(real_t)*M2Ltargets_idx_size*n3, hipMemcpyDeviceToHost); 
    hipFree(d_dnCheck);
    hipFree(d_dw_equiv_fft);
    return dnCheck;
  }

  hipfftComplex *HadmardGPU(std::vector<int> &M2Ltargets_idx, std::vector<int> &M2LRelPos_start_idx, std::vector<int> &index_in_up_equiv_fft, std::vector<int> &M2LRelPoss, RealVec mat_M2L_Helper, int n3_, hipfftComplex *d_up_equiv_fft) {
    int BLOCKS = M2Ltargets_idx.size();
    int THREADS = n3_;

    int *d_M2Ltargets_idx, *d_M2LRelPos_start_idx, *d_index_in_up_equiv_fft, *d_M2LRelPoss;
    real_t *d_mat_M2L_Helper;
    hipfftComplex *d_dw_equiv_fft;
    hipMalloc(&d_M2Ltargets_idx, sizeof(int)*M2Ltargets_idx.size());
    hipMalloc(&d_M2LRelPos_start_idx, sizeof(int)*M2LRelPos_start_idx.size());
    hipMalloc(&d_index_in_up_equiv_fft, sizeof(int)*index_in_up_equiv_fft.size());
    hipMalloc(&d_M2LRelPoss, sizeof(int)*M2LRelPoss.size());
    hipMalloc(&d_dw_equiv_fft, sizeof(hipfftComplex)*M2Ltargets_idx.size()*n3_);
    hipMalloc(&d_mat_M2L_Helper, sizeof(real_t)*mat_M2L_Helper.size());

    hipMemcpy(d_M2Ltargets_idx, &M2Ltargets_idx[0], sizeof(int)*M2Ltargets_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_M2LRelPos_start_idx, &M2LRelPos_start_idx[0], sizeof(int)*M2LRelPos_start_idx.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_index_in_up_equiv_fft, &index_in_up_equiv_fft[0], sizeof(int)*index_in_up_equiv_fft.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_M2LRelPoss, &M2LRelPoss[0], sizeof(int)*M2LRelPoss.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_mat_M2L_Helper, &mat_M2L_Helper[0], sizeof(real_t)*mat_M2L_Helper.size(), hipMemcpyHostToDevice);    
    hadmard_kernel<<<BLOCKS, THREADS>>>(d_M2Ltargets_idx, d_up_equiv_fft, d_dw_equiv_fft, d_M2LRelPos_start_idx, d_index_in_up_equiv_fft, d_M2LRelPoss, d_mat_M2L_Helper, n3_, BLOCKS);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipFree(d_M2Ltargets_idx);
    hipFree(d_M2LRelPos_start_idx);
    hipFree(d_index_in_up_equiv_fft);
    hipFree(d_M2LRelPoss);
    hipFree(d_up_equiv_fft);
    hipFree(d_mat_M2L_Helper);
    return &d_dw_equiv_fft[0];
  }
  
  std::vector<real_t> M2LGPU(std::vector<int> &M2Ltargets_idx, std::vector<int> &M2LRelPos_start_idx, std::vector<int> &index_in_up_equiv_fft, std::vector<int> &M2LRelPoss, RealVec mat_M2L_Helper, int n3_, AlignedVec &up_equiv, int M2Lsources_idx_size) {
    hipfftComplex *d_up_equiv_fft = FFT_UpEquiv_GPU(M2Lsources_idx_size, up_equiv);
    hipfftComplex *d_dw_equiv_fft = HadmardGPU(M2Ltargets_idx, M2LRelPos_start_idx, index_in_up_equiv_fft, M2LRelPoss, mat_M2L_Helper, n3_, d_up_equiv_fft);
    return FFT_Check2Equiv_GPU(d_dw_equiv_fft, M2Ltargets_idx.size());
  }
}
